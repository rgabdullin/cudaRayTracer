#include <cstdio>
#include "Objects.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CHECK_CUDA_ERRORS {hipError_t x = hipGetLastError(); if ((x) != hipSuccess) { printf("Error: %s\n", hipGetErrorString(x)); system("pause"); return 1; }}

int main(void) {
	Object** dev_array;
	int n = 8;
	hipMalloc((void**)&dev_array, sizeof(Object*) * n);
	test::GPUCreateTestObjectArray<<<1,n>>>(dev_array);
	hipDeviceSynchronize();
	CHECK_CUDA_ERRORS;
	
	test::kernel <<< 1, n >>> (dev_array);
	hipDeviceSynchronize();
	CHECK_CUDA_ERRORS;

	test::GPUDestroyTestObjectArray<<<1,n>>>(dev_array);
	hipDeviceSynchronize();
	CHECK_CUDA_ERRORS;

	hipFree(dev_array);
	CHECK_CUDA_ERRORS;

	system("pause");
	return 0;
}