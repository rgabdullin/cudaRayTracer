#include "hip/hip_runtime.h"
#define SYNC_AND_CHECK_CUDA_ERRORS {hipDeviceSynchronize(); hipError_t x = hipGetLastError(); if ((x) != hipSuccess) { printf("Error: %s\n", hipGetErrorString(x)); system("pause"); exit(1); }}

#include "RegularSampler.h"

#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

__global__
extern void kernel(void);

int main()
{
	kernel <<< 1, 1 >>> ();
	SYNC_AND_CHECK_CUDA_ERRORS;

	system("pause");
	return 0;
}