#include "hip/hip_runtime.h"
#ifndef SYNC_AND_CHECK_CUDA_ERRORS
#define SYNC_AND_CHECK_CUDA_ERRORS {hipStreamSynchronize(0); hipError_t x = hipGetLastError(); if ((x) != hipSuccess) { printf("Error: %s\n", hipGetErrorString(x)); fclose(stdout); exit(1); }}
#endif


#include <cstdio>
#include <cstdlib>
#include <ctime> 
#include <string>

#include "World.h"
#include "utils.hpp"

int main(void) {
	time_t tm; time(&tm);

	freopen("output.log", "a", stdout);

	time_t start_time; time(&start_time);

	clock_t start_clock = clock();

	struct tm* s_time = localtime(&start_time);

	printf("================= Runnning. time = %04d/%02d/%02d %02d:%02d:%02d =================\n", s_time->tm_year + 1900, s_time->tm_mon + 1, s_time->tm_mday, s_time->tm_hour, s_time->tm_min, s_time->tm_sec);

	CUDAInfo();

	World* w;
	hipMallocManaged(&w, sizeof(World));

	w->init(1 * make_int2(256, 128), 4, 64);

	printf("Building scene\n\t");
	w->build_scene();
	printf("\tOK\n");

	//Tracing
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start, 0);
	
	w->render_scene(64 * 64);

	hipEventRecord(stop, 0); hipStreamSynchronize(0);

	float _time;
	hipEventElapsedTime(&_time, start, stop);
	printf("Tracing time: %.2f ms\n\n", _time);
	
	//Saving image
	w->save_image("./kek.bmp");
	w->clear();
	
	hipFree(w);
	
	clock_t end_clock = clock();
	printf("TIME ELAPSED: %lf\n", (end_clock - start_clock) / 1000.0);

	fclose(stdout);

	return 0;
}