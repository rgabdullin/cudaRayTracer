#include "hip/hip_runtime.h"
#define SYNC_AND_CHECK_CUDA_ERRORS {hipDeviceSynchronize(); hipError_t x = hipGetLastError(); if ((x) != hipSuccess) { printf("Error: %s\n", hipGetErrorString(x)); system("pause"); exit(1); }}

#include <cstdio>
#include <cstdlib>

#include "World.h"
#include "utils.hpp"

int main(void) {
	CUDAInfo();

	World* w;
	hipMallocManaged(&w, sizeof(World));

	w->init(2 * make_int2(1024, 512), 4);

	printf("Building scene\n\t");
	w->build_scene();
	printf("\tOK\n");

	//Tracing
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start, 0);
	
	w->render_scene();

	hipEventRecord(stop, 0); hipStreamSynchronize(0);

	float time;
	hipEventElapsedTime(&time, start, stop);
	printf("Tracing time: %.2f ms\n\n", time);
	
	//Saving image
	printf("Saving image\n\t");
	w->save_image("./kek.bmp");
	printf("OK\n");
	w->clear();
	
	hipFree(w);

	system("pause");
	return 0;
}